#include "hip/hip_runtime.h"
/*
@author: Deniz Uzel
*/

#include <cstdio>
#include "util.h"
#include <cmath>


#define MAX_NO_THREADS_IN_A_BLOCK 1024 // Tesla K4 GoogleColab
#define MAX_WIDTH 32

#define KERNEL_NUMBER_OF_ELEMENTS 100


__constant__ int d_kernel[KERNEL_NUMBER_OF_ELEMENTS];


__global__ void conv(int* d_in,  int* d_out,
                     int kernel_size,
                     int num_rows, int num_cols,
                     int extended_num_rows, int extended_num_cols,
                     int shared_memory_size)
{

    // amount of padding for each side
    int p = kernel_size / 2;

    // number of thread num in one side
    int BLOCK_WIDTH = MAX_WIDTH;

    // initialize tile array
    __shared__ int tile[MAX_NO_THREADS_IN_A_BLOCK];

    int inner_tile_width = BLOCK_WIDTH - 2*p;

    int ty = threadIdx.y;
    int tx = threadIdx.x;

    int row_out = blockIdx.y*inner_tile_width + ty;
    int col_out = blockIdx.x*inner_tile_width + tx;

    int row_start = row_out;
    int col_start = col_out;

    if ((row_start < extended_num_rows) && (col_start < extended_num_cols)
        && row_start >= 0 && col_start >=0){
        tile[ty*BLOCK_WIDTH + tx] = d_in[row_start*extended_num_cols + col_start];
    } else {
        tile[ty*BLOCK_WIDTH + tx] = 0;
    }


    __syncthreads();

    // checks if the thread is valid for the convolution
    if ((ty < inner_tile_width) && (tx < inner_tile_width)){

        // convolution output value
        int sum_val = 0;

        for (int i = ty; i < (ty+kernel_size); i++){
            for (int j = tx; j < (tx+kernel_size); j++){

                int val_in = tile[i*BLOCK_WIDTH + j];
                int val_ker = d_kernel[(i-ty)*kernel_size + j-tx];

                sum_val += val_in * val_ker;
            }
        }

        // normalize the output value
        int kernel_sum = 0;

        for (int i=0; i < kernel_size*kernel_size; i++){
                kernel_sum += d_kernel[i];
        }

        double normalized_val = (double)sum_val / kernel_sum;

        if (row_out < num_rows && col_out < num_cols){
            d_out[row_out*num_cols + col_out] = (int)normalized_val;
        }
    }
}


int main(int argc, char* argv[]) {

    int num_rows, num_cols, kernel_size;

    // load image and kernel
    int** img = read_pgm_file(argv[1], &num_rows, &num_cols);
    int** kernel = read_pgm_file(argv[2], &kernel_size, &kernel_size);

    // extend the image
    int* h_extendedImg;

    int extend_amount = kernel_size / 2;
    int extended_num_rows = num_rows + extend_amount*2;
    int extended_num_cols = num_cols + extend_amount*2;

    h_extendedImg = extend_edges(img, num_rows-1, num_cols-1, extend_amount);


    // CUDA Part

    const int INPUT_ARRAY_SIZE = extended_num_rows * extended_num_cols;
    const int INPUT_ARRAY_BYTES = INPUT_ARRAY_SIZE * sizeof(int);

    const int OUTPUT_ARRAY_SIZE = num_rows * num_cols;
    const int OUTPUT_ARRAY_BYTES = OUTPUT_ARRAY_SIZE * sizeof(int);

    const int KERNEL_ARRAY_SIZE = kernel_size * kernel_size;
    const int KERNEL_ARRAY_BYTES = KERNEL_ARRAY_SIZE * sizeof(int);


    int* h_kernel = convert_kernel_2d_to_1d(kernel, kernel_size, kernel_size);
    int* h_out = (int*)malloc(OUTPUT_ARRAY_BYTES);

    int* d_in;
    int* d_out;

    hipMalloc((void**)&d_in, INPUT_ARRAY_BYTES);
    hipMalloc((void**)&d_out, OUTPUT_ARRAY_BYTES);
    hipMalloc((void**)&d_kernel, KERNEL_ARRAY_BYTES);

    hipMemcpy(d_in, h_extendedImg, INPUT_ARRAY_BYTES, hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(d_kernel), h_kernel, KERNEL_ARRAY_BYTES);

    // amount of padding for each side
    int p = kernel_size / 2;

    // dimensions
    int thread = MAX_WIDTH; // MAX_NO_THREADS_IN_A_BLOCK >> 5;  // 1024/32 = 32
    int block_width = ceil((double)extended_num_rows / (thread-2*p));  // +1 is for defensive coding
    int block_height = ceil((double)extended_num_cols / (thread-2*p));

    dim3 threads_per_block(thread, thread);
    dim3 blocks_per_grid(block_width, block_height);


    // Total amount of shared memory per block: 49152 bytes
    const int SHARED_MEMORY_SIZE = MAX_NO_THREADS_IN_A_BLOCK;
    const int SHARED_MEMORY_BYTES = SHARED_MEMORY_SIZE * sizeof(int);

    // To Do: Conv KERNEL HERE
    conv <<<blocks_per_grid, \
            threads_per_block, \
            SHARED_MEMORY_BYTES >>> (d_in, d_out,
                                     kernel_size,
                                     num_rows, num_cols,
                                     extended_num_rows, extended_num_cols,
                                     SHARED_MEMORY_SIZE);

    hipMemcpy(h_out, d_out, OUTPUT_ARRAY_BYTES, hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_kernel);

    // write the output to file
    char* output_file_name = argv[3];
    writeFile_1d(output_file_name, h_out, num_rows, num_cols);

    free(h_extendedImg);
    free(h_kernel);
    free(h_out);

    dealloc_2d_matrix(img, num_rows, num_cols);
    dealloc_2d_matrix(kernel, kernel_size, kernel_size);
}